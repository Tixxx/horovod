#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <time.h>

#define THREADS_PER_BLOCK 64

template<typename T, typename TACC>
__global__
void CudaDotProductKernel(int count, const T* a, const T* b, TACC* out_normsq_a, TACC* out_normsq_b, TACC* out_dot) {
	__shared__ TACC normsq_a[THREADS_PER_BLOCK];
	__shared__ TACC normsq_b[THREADS_PER_BLOCK];
	__shared__ TACC dot[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < count){
		normsq_a[threadIdx.x] = (TACC) a[index] * (TACC) a[index];
		normsq_b[threadIdx.x] = (TACC) b[index] * (TACC) b[index];
		dot[threadIdx.x]      = (TACC) a[index] * (TACC) b[index];
	}
	__syncthreads();
	if (0 == threadIdx.x) {
		TACC normsq_a_sum = 0;
		TACC normsq_b_sum = 0;
		TACC dot_sum = 0;
		for(int i = 0; i < THREADS_PER_BLOCK; i++){
			if (i + blockIdx.x * blockDim.x < count){
				normsq_a_sum += normsq_a[i];
				normsq_b_sum += normsq_b[i];
				dot_sum += dot[i];
			}
		}
		atomicAdd(out_normsq_a, normsq_a_sum);
		atomicAdd(out_normsq_b, normsq_b_sum);
		atomicAdd(out_dot, dot_sum);
	}
}

template<typename T, typename TACC>
__global__
void CudaScaleAddKernel(int count, T* a, const T* b, TACC a_coeff, TACC b_coeff) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (count > index){
		a[index] = (T) ((TACC) a[index] * a_coeff + (TACC) b[index] * b_coeff);
	}
}

void CudaDotProductImpl(int count, const double* device_a, const double* device_b, 
						double* device_normsq_a, double* device_normsq_b, double* device_dot) {
	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
}

void CudaDotProductImpl(int count, const float* device_a, const float* device_b, 
						double* device_normsq_a, double* device_normsq_b, double* device_dot) {
	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
}

void CudaDotProductImpl(int count, const uint16_t* device_a, const uint16_t* device_b, 
						double* device_normsq_a, double* device_normsq_b, double* device_dot) {
	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, (__half*)device_a, (__half*)device_b, device_normsq_a, device_normsq_b, device_dot);
}

void CudaScaleAddImpl(int count, double* a_device, const double* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, float* a_device, const float* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, uint16_t* a_device, const uint16_t* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, (__half*)a_device, (__half*)b_device,
		host_a_coeff, host_b_coeff);
}


template<typename T, typename TACC>
void psl_cuda_reduction(int count, T* a, T* b, TACC* out_normsq_a, TACC* out_normsq_b, TACC* out_dot){
	TACC normsq_a = 0.f;
	TACC normsq_b = 0.f;
	TACC dot = 0.f;
	hipMemcpy(out_normsq_a, &normsq_a, sizeof(TACC), hipMemcpyHostToDevice);
	hipMemcpy(out_normsq_b, &normsq_b, sizeof(TACC), hipMemcpyHostToDevice);
	hipMemcpy(out_dot, &dot, sizeof(TACC), hipMemcpyHostToDevice);
	normsq_and_dot<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a, b, out_normsq_a, out_normsq_b, out_dot);
	hipMemcpy(&normsq_a, out_normsq_a, sizeof(TACC), hipMemcpyDeviceToHost);
	hipMemcpy(&normsq_b, out_normsq_b, sizeof(TACC), hipMemcpyDeviceToHost);
	hipMemcpy(&dot, out_dot, sizeof(TACC), hipMemcpyDeviceToHost);
	TACC a_coeff = 1;
	TACC b_coeff = 1;           
	if (normsq_a != 0) 
		a_coeff = 1.0 - dot / normsq_a * 0.5;                                                                                                                                                                                                                      
	if (normsq_b != 0)
		b_coeff = 1.0 - dot / normsq_b * 0.5;

	saxpy<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a, b, a_coeff, b_coeff);
}