
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <time.h>
#include <stdint.h>

#define THREADS_PER_BLOCK 64

template<typename T, typename TACC>
__global__
void CudaDotProductKernel(int count, const T* a, const T* b, TACC* out_normsq_a, TACC* out_normsq_b, TACC* out_dot) {
	__shared__ TACC normsq_a[THREADS_PER_BLOCK];
	__shared__ TACC normsq_b[THREADS_PER_BLOCK];
	__shared__ TACC dot[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < count){
		normsq_a[threadIdx.x] = (TACC) a[index] * (TACC) a[index];
		normsq_b[threadIdx.x] = (TACC) b[index] * (TACC) b[index];
		dot[threadIdx.x]      = (TACC) a[index] * (TACC) b[index];
	}
	__syncthreads();
	if (0 == threadIdx.x) {
		TACC normsq_a_sum = 0;
		TACC normsq_b_sum = 0;
		TACC dot_sum = 0;
		for(int i = 0; i < THREADS_PER_BLOCK; i++){
			if (i + blockIdx.x * blockDim.x < count){
				normsq_a_sum += normsq_a[i];
				normsq_b_sum += normsq_b[i];
				dot_sum += dot[i];
			}
		}
		atomicAdd(out_normsq_a, normsq_a_sum);
		atomicAdd(out_normsq_b, normsq_b_sum);
		atomicAdd(out_dot, dot_sum);
	}
}

template<typename T, typename TACC>
__global__
void CudaScaleAddKernel(int count, T* a, const T* b, TACC a_coeff, TACC b_coeff) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (count > index){
		a[index] = (T) ((TACC) a[index] * a_coeff + (TACC) b[index] * b_coeff);
	}
}

void CudaDotProductImpl(int count, const double* device_a, const double* device_b, 
	double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice);

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
	hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost);

}

void CudaDotProductImpl(int count, const float* device_a, const float* device_b, 
						double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice);

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
	hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost);

}

void CudaDotProductImpl(int count, const uint16_t* device_a, const uint16_t* device_b, 
	double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice);

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, (__half*) device_a, (__half*) device_b, device_normsq_a, device_normsq_b, device_dot);
	hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost);

}

void CudaScaleAddImpl(int count, double* a_device, const double* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, float* a_device, const float* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, uint16_t* a_device, const uint16_t* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, (__half*)a_device, (__half*)b_device,
		host_a_coeff, host_b_coeff);
}