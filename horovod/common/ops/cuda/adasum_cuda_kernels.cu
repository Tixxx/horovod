// Copyright 2019 Microsoft. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// =============================================================================


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <time.h>
#include <stdint.h>

#define THREADS_PER_BLOCK 64

template<typename T, typename TACC>
__global__
void CudaDotProductKernel(int count, const T* a, const T* b, TACC* out_normsq_a, TACC* out_normsq_b, TACC* out_dot) {
	__shared__ TACC normsq_a[THREADS_PER_BLOCK];
	__shared__ TACC normsq_b[THREADS_PER_BLOCK];
	__shared__ TACC dot[THREADS_PER_BLOCK];
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < count){
		normsq_a[threadIdx.x] = (TACC) a[index] * (TACC) a[index];
		normsq_b[threadIdx.x] = (TACC) b[index] * (TACC) b[index];
		dot[threadIdx.x]      = (TACC) a[index] * (TACC) b[index];
	}
	__syncthreads();
	if (0 == threadIdx.x) {
		TACC normsq_a_sum = 0;
		TACC normsq_b_sum = 0;
		TACC dot_sum = 0;
		for(int i = 0; i < THREADS_PER_BLOCK; i++){
			if (i + blockIdx.x * blockDim.x < count){
				normsq_a_sum += normsq_a[i];
				normsq_b_sum += normsq_b[i];
				dot_sum += dot[i];
			}
		}
		atomicAdd(out_normsq_a, normsq_a_sum);
		atomicAdd(out_normsq_b, normsq_b_sum);
		atomicAdd(out_dot, dot_sum);
	}
}

template<typename T, typename TACC>
__global__
void CudaScaleAddKernel(int count, T* a, const T* b, TACC a_coeff, TACC b_coeff) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (count > index){
		a[index] = (T) ((TACC) a[index] * a_coeff + (TACC) b[index] * b_coeff);
	}
}

template<typename T>
__global__
void ConvertToFloat(int count, T* a, float* b) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (count > index){
		b[index] = (float) a[index];
	}
}

void ErrorCheck(std::string op_name, hipError_t cuda_result) {
	if (cuda_result != hipSuccess) {
		throw std::logic_error(std::string(op_name) + " failed: " + hipGetErrorString(cuda_result));
	}
}

void CudaDotProductImpl(int count, const double* device_a, const double* device_b, 
	double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice));

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
	ErrorCheck("CudaDotProductKernel(double)", hipGetLastError());
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost));

}

void CudaDotProductImpl(int count, const float* device_a, const float* device_b, 
						double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice));

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, device_a, device_b, device_normsq_a, device_normsq_b, device_dot);
	ErrorCheck("CudaDotProductKernel(float)", hipGetLastError());
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost));

}

void CudaDotProductImpl(int count, const uint16_t* device_a, const uint16_t* device_b, 
	double* device_normsq_a, double* device_normsq_b, double* device_dot, double& host_normsq_a, double& host_normsq_b, double& host_dot) {
	
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_a, &host_normsq_a, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_normsq_b, &host_normsq_b, sizeof(double), hipMemcpyHostToDevice));
	ErrorCheck("hipMemcpy", hipMemcpy(device_dot, &host_dot, sizeof(double), hipMemcpyHostToDevice));

	CudaDotProductKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,
		THREADS_PER_BLOCK>>>(count, (__half*) device_a, (__half*) device_b, device_normsq_a, device_normsq_b, device_dot);
	ErrorCheck("CudaDotProductKernel(fp16)", hipGetLastError());
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_a, device_normsq_a, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_normsq_b, device_normsq_b, sizeof(double), hipMemcpyDeviceToHost));
	ErrorCheck("hipMemcpy", hipMemcpy(&host_dot, device_dot, sizeof(double), hipMemcpyDeviceToHost));

}

void CudaScaleAddImpl(int count, double* a_device, const double* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, float* a_device, const float* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, a_device, b_device,
		host_a_coeff, host_b_coeff);
}

void CudaScaleAddImpl(int count, uint16_t* a_device, const uint16_t* b_device, double host_a_coeff, double host_b_coeff) {
	CudaScaleAddKernel<<<(count+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(count, (__half*)a_device, (__half*)b_device,
		host_a_coeff, host_b_coeff);
}
